#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CEIL(a,b) ((a+b-1)/(b))

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}




__global__ void sigmoid(float* x, float* y, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        y[idx] = 1.0f / (1.0f + expf(-x[idx]));
    }
}





int main (void) {
    constexpr int N = 333;
    float* x_h = (float*)malloc(N * sizeof(float));
    float* y_h = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i ++) {
        x_h[i] = 1.0f / (1 + i);
    }

    
    float* x_d = nullptr;
    float* y_d = nullptr;
    CudaSafeCall(hipMalloc(&x_d, N * sizeof(float)));
    CudaSafeCall(hipMalloc(&y_d, N *sizeof(float)));
    hipMemcpy(x_d, x_h, N * sizeof(float), hipMemcpyHostToDevice);
    

    int block_size = 256;
    int grid_size = CEIL(N, block_size);
    sigmoid<<<grid_size, block_size>>>(x_d, y_d, N);
    CudaCheckError();

    CudaSafeCall(hipMemcpy(y_h, y_d, N * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i ++) {
        printf("%.2f %s", y_h[i], (i+1)%10 == 0? "\n":"");
    }


    



    return 0;
}